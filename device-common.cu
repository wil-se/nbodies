#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "common.h"
#include "device-common.h"
#include <stdio.h>

__managed__ double *x_d, *y_d, *z_d, *new_x_d, *new_y_d, *new_z_d, *new_sx_d, *new_sy_d, *new_sz_d, *mass_d, *sx_d, *sy_d, *sz_d;
__managed__ int n_d;

void set_memory_cuda(){
	//hipMalloc((void**)&n_d, sizeof(int));
    hipMalloc((void**)&x_d, sizeof(double)*n);
    hipMalloc((void**)&y_d, sizeof(double)*n);
    hipMalloc((void**)&z_d, sizeof(double)*n);
    hipMalloc((void**)&mass_d, sizeof(double)*n);
    hipMalloc((void**)&sx_d, sizeof(double)*n);
    hipMalloc((void**)&sy_d, sizeof(double)*n);
    hipMalloc((void**)&sz_d, sizeof(double)*n);
    
    hipMemcpy(&n_d, &n, sizeof(int), hipMemcpyHostToDevice);
    
    hipMemcpy(x_d, x, sizeof(double)*n, hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, sizeof(double)*n, hipMemcpyHostToDevice);
    hipMemcpy(z_d, z, sizeof(double)*n, hipMemcpyHostToDevice);
    hipMemcpy(mass_d, mass, sizeof(double)*n, hipMemcpyHostToDevice);
    hipMemcpy(sx_d, sx, sizeof(double)*n, hipMemcpyHostToDevice);
    hipMemcpy(sy_d, sy, sizeof(double)*n, hipMemcpyHostToDevice);
    hipMemcpy(sz_d, sz, sizeof(double)*n, hipMemcpyHostToDevice);


    hipDeviceSynchronize();
}

void swap_memory(){
	hipMemcpy(x, x_d, sizeof(double)*n, hipMemcpyDeviceToHost);
    hipMemcpy(y, y_d, sizeof(double)*n, hipMemcpyDeviceToHost);
    hipMemcpy(z, z_d, sizeof(double)*n, hipMemcpyDeviceToHost);
    hipMemcpy(mass, mass_d, sizeof(double)*n, hipMemcpyDeviceToHost);
    hipMemcpy(sx, sx_d, sizeof(double)*n, hipMemcpyDeviceToHost);
    hipMemcpy(sy, sy_d, sizeof(double)*n, hipMemcpyDeviceToHost);
    hipMemcpy(sz, sz_d, sizeof(double)*n, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
}

void free_memory_cuda(){
    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
    hipFree(mass_d);
    hipFree(sx_d);
    hipFree(sy_d);
    hipFree(sz_d);
}

__global__ void set_new_memory_cuda(){
	hipMalloc((void**)&new_x_d, sizeof(double)*n_d);
	hipMalloc((void**)&new_y_d, sizeof(double)*n_d);
	hipMalloc((void**)&new_z_d, sizeof(double)*n_d);
	hipMalloc((void**)&new_sx_d, sizeof(double)*n_d);
	hipMalloc((void**)&new_sy_d, sizeof(double)*n_d);
	hipMalloc((void**)&new_sz_d, sizeof(double)*n_d);
}

__global__ void free_new_memory_cuda(){
	hipFree(new_x_d);
	hipFree(new_y_d);
	hipFree(new_z_d);
	hipFree(new_sx_d);
	hipFree(new_sy_d);
	hipFree(new_sz_d);
}

__global__ void set_new_vectors_cuda(){
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    new_x_d[i] = x_d[i];
    new_y_d[i] = y_d[i];
    new_z_d[i] = z_d[i];
    new_sx_d[i] = sx_d[i];
    new_sy_d[i] = sy_d[i];
    new_sz_d[i] = sz_d[i];      
}

__global__ void set_vectors_cuda(){
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    x_d[i] = new_x_d[i];
    y_d[i] = new_y_d[i];
    z_d[i] = new_z_d[i];
    sx_d[i] = new_sx_d[i];
    sy_d[i] = new_sy_d[i];
    sz_d[i] = new_sz_d[i];      
}

