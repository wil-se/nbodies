#include <stdio.h>
#include <stdlib.h> 
#include <hip/hip_runtime.h>
#include "common.h"
#include "render.h"
#include "device-common.h"
#include "cuda-exhaustive.h"

int main(int argc, char** argv) {
  set_memory();
  set_memory_cuda();
  
  // hipDeviceSynchronize();
  // render_sequential_barneshut(argc, argv);
  // render_sequential_exhaustive(argc, argv);
  render_cuda_exhaustive(argc, argv);
  hipDeviceSynchronize();
  
  free_memory();
  return 0;
}