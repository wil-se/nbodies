
#include <hip/hip_runtime.h>


float4 *octree;
int *child_locks; 

// TODO isolare in .h esterno
#define GPU_CHECK(call) { checkCudaError(call, __FILE__, __LINE__); }

inline void checkCudaError( hipError_t code, const char* s, int line) {
        if (code != hipSuccess) {
                fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), s, line);
                printf("[CUDA ERROR} %s \n", hipGetErrorString(code));
                printf("REASON: %s\n", hipGetErrorName(code));
                exit(1);
        }

}

__host__ __device__ 
void init_octree(int nbodies) {
        size_t size = nbodies*8;
        GPU_CHECK(hipMalloc((void**)&octree, sizeof(float4)*size));
        GPU_CHECK(hipMalloc((void**)&child_locks, sizeof(int)*size));
}

__device__
int acquire_lock() {
        

}
