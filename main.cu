#include <stdio.h>
#include <stdlib.h> 
#include <hip/hip_runtime.h>
#include "common.h"
#include "render.h"
#include "device-common.h"
#include "cuda-exhaustive.h"
#include "cuda-barneshut.h"
#include "openmp.h"

int omp_num_th = 1;

int main(int argc, char** argv) {
        if (argc > 1) {
                int t = atoi(argv[1]);
                if (t > 0) omp_num_th = t;
        } 
        set_memory();
        // set_memory_cuda();

        // hipDeviceSynchronize();
        // render_sequential_barneshut(argc, argv);
        // render_sequential_exhaustive(argc, argv);
        // render_cuda_exhaustive(argc, argv);
        // render_cuda_barneshut(argc, argv);
        // compute_barneshut_forces_cuda<<<1,1>>>();
        // hipDeviceSynchronize();


        // OPENMP
        //exhaustive_openmp();
        barneshut_openmp();

        free_memory();
        return 0;
}
